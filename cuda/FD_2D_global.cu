#include "hip/hip_runtime.h"
/*** Calculating a derivative with CD ***/
#include <iostream>
#include <fstream>
#include <cmath>
#include <sys/time.h>

void checkErrors(char *label)
{
// we need to synchronise first to catch errors due to
// asynchroneous operations that would otherwise
// potentially go unnoticed
hipError_t err;
err = hipDeviceSynchronize();
if (err != hipSuccess)
{
char *e = (char*) hipGetErrorString(err);
fprintf(stderr, "CUDA Error: %s (at %s)\n", e, label);
}
err = hipGetLastError();
if (err != hipSuccess)
{
char *e = (char*) hipGetErrorString(err);
fprintf(stderr, "CUDA Error: %s (at %s)\n", e, label);
}
}
	
double get_time() 
{  struct timeval tim;
  hipDeviceSynchronize();
  gettimeofday(&tim, NULL);
  return (double) tim.tv_sec+(tim.tv_usec/1000000.0);
}

__global__ void copy_array(float *u, float *u_prev, int N, int BSZ)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	int I = blockIdx.y*BSZ*N + blockIdx.x*BSZ + j*N + i;
	if (I>=N*N){return;}	
	u_prev[I] = u[I];

}

// GPU kernel
__global__ void update (float *u, float *u_prev, int N, float h, float dt, float alpha, int BSZ)
{
	// Setting up indices
	int i = threadIdx.x;
	int j = threadIdx.y;
	int I = blockIdx.y*BSZ*N + blockIdx.x*BSZ + j*N + i;
	
	if (I>=N*N){return;}	
	//if (()>=N || j>){return;}	

	
	// if not boundary do
	if ( (I>N) && (I< N*N-1-N) && (I%N!=0) && (I%N!=N-1)) 
	{	u[I] = u_prev[I] + alpha*dt/(h*h) * (u_prev[I+1] + u_prev[I-1] + u_prev[I+N] + u_prev[I-N] - 4*u_prev[I]);
	}
	
	// Boundary conditions are automatically imposed
	// as we don't touch boundaries
}

int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("Invalid number of arguments");
        return;
    }
	// Allocate in CPU
	int N = atoi(argv[1]);
	int BLOCKSIZE = atoi(argv[2]);

	hipSetDevice(0);

	float xmin 	= 0.0f;
	float xmax 	= 3.5f;
	float ymin 	= 0.0f;
	//float ymax 	= 2.0f;
	float h   	= (xmax-xmin)/(N-1);
	float dt	= 0.00001f;	
	float alpha	= 0.645f;
	float time 	= 0.4f;

	//int steps = ceil(time/dt);
	int steps = 100;
	int I;

	float *x  	= new float[N*N]; 
	float *y  	= new float[N*N]; 
	float *u  	= new float[N*N];
	float *u_prev  	= new float[N*N];


	// Generate mesh and intial condition
	for (int j=0; j<N; j++)
	{	for (int i=0; i<N; i++)
		{	I = N*j + i;
			x[I] = xmin + h*i;
			y[I] = ymin + h*j;
			u[I] = 0.0f;
			if ( (i==0) || (j==0)) 
				{u[I] = 200.0f;}
		}
	}

	// Allocate in GPU
	float *u_d, *u_prev_d;
	
	hipMalloc( (void**) &u_d, N*N*sizeof(float));
	hipMalloc( (void**) &u_prev_d, N*N*sizeof(float));

	// Copy to GPU
	hipMemcpy(u_d, u, N*N*sizeof(float), hipMemcpyHostToDevice);

	// Loop 
	dim3 dimGrid(int((N-0.5)/BLOCKSIZE)+1, int((N-0.5)/BLOCKSIZE)+1);
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	double start = get_time();
	for (int t=0; t<steps; t++)
	{	copy_array <<<dimGrid, dimBlock>>> (u_d, u_prev_d, N, BLOCKSIZE);
		update <<<dimGrid, dimBlock>>> (u_d, u_prev_d, N, h, dt, alpha, BLOCKSIZE);

	}
	double stop = get_time();
	checkErrors("update");
	
	double elapsed = stop - start;
	// std::cout<<"time = "<<elapsed<<std::endl;
    // std::cout << N << "," << BLOCKSIZE << "," << elapsed << std::endl;
	// Copy result back to host
	hipMemcpy(u, u_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
    /*
	std::ofstream temperature("temperature_global.txt");
	for (int j=0; j<N; j++)
	{	for (int i=0; i<N; i++)
		{	I = N*j + i;
		//	std::cout<<u[I]<<"\t";
			temperature<<x[I]<<"\t"<<y[I]<<"\t"<<u[I]<<std::endl;
		}
		temperature<<"\n";
		//std::cout<<std::endl;
	}

	temperature.close();
    */
	// Free device
	hipFree(u_d);
	hipFree(u_prev_d);
}
